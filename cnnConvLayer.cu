#include "hip/hip_runtime.h"
// This program executes a typical convolutional layer in regular CNNs
//in CSR format
//in COO format

#include <iostream>
#include "cnnConvLayer.h"
#include <stdio.h>
#include <unistd.h>
using namespace std;

#define xDim 512
#define yDim 32
#define zDim 32

#define xThreadDim 4
#define yThreadDim 16
#define zThreadDim 16

#define Pool_xDim 512
#define Pool_yDim 16
#define Pool_zDim 16


int outputsize = 512*16*16;
int Outputsize = xDim*yDim*zDim;

int *devoutNeu;
int *devPooling;
short *devFilt;
short *devinNeu;
int *devGlobalBarrier;

/*COO Format*/
short *devfiltCooNNZ;
short *devfiltCooData;
short *devfiltCooRow;
short *devfiltCooCol;

short *devinNeuCooNNZ;
short *devinNeuCooData;
short *devinNeuCooRow;
short *devinNeuCooCol;


int *outResult = new int[outputsize]();
int *outResult_neu = new int[Outputsize]();
int *outGlobalBarrier = new int[Outputsize]();


// This is the CPU version, please don't modify it
void convLayerCPU()
{
	// declarations for bunch of indexing parameters
	int fn, sli, fmy, fmx, y, x;
	int sum, ifmy, ifmx, ofmy, ofmx;
	int filtIdx, inNeuIdx, outNeuIdx, outIdx;
	int filtVol = FMDEPTH * FILTSIZE * FILTSIZE;
	int filtArea = FILTSIZE * FILTSIZE;
	int fmArea = FMSIZE *FMSIZE;
	int outArea = FMSIZE/2 * FMSIZE/2;


	cout << "convolutioning..." << endl;

	// Convolution
	for(fn = 0; fn < FILTNUM; fn++) //512
	{
		for(fmy = 0; fmy < FMSIZE; fmy += STRIDE) //32
		{
			for(fmx = 0; fmx < FMSIZE; fmx += STRIDE)  //32
			{
				


				sum = 0;
				for(sli = 0; sli < FMDEPTH; sli++)  //512
				{
					for(y = 0; y < FILTSIZE; y++)  //3
					{
						for(x = 0; x < FILTSIZE; x++)  //3
						{
							ifmy = fmy - FILTSIZE / 2 + y;		//no dependancy
							ifmx = fmx - FILTSIZE / 2 + x;		//no dependancy
							filtIdx = (fn * filtVol) + (sli * filtArea) + (y * FILTSIZE) + x;	//no dependancy
							inNeuIdx = sli*fmArea + ifmy*FMSIZE + ifmx;							//no dependancy
							if(ifmy >= 0 && ifmy < FMSIZE && ifmx >= 0 && ifmx < FMSIZE)
								sum += filt[filtIdx] * inNeu[inNeuIdx];
							//if(fn == 0 && fmx == 0 && fmy == 0 && sli <10)
							//	printf("filt[%d] = %d\n",filtIdx,filt[filtIdx] );
						}
					}
				}



				// Activation - ReLU
				outNeuIdx = fn*fmArea + fmy*FMSIZE + fmx;
				if(sum <= 0)
					outNeu[outNeuIdx] = 0;
				else
					outNeu[outNeuIdx] = sum;
			}
		}
	}


 	cout << "Pooling....." << endl;
	// Max Pooling with Window Size 2x2
	int max, tmpVal;
	for(sli = 0; sli < FILTNUM; sli++)
	{
		for(fmy = 0; fmy < FMSIZE/2 ; fmy += 1)
		{
			for(fmx = 0; fmx < FMSIZE/2 ; fmx += 1)
			{
				outNeuIdx = sli*fmArea + fmy*2*FMSIZE + fmx*2;
				max = outNeu[outNeuIdx];
				for(y = 0; y < 2; y++)
				{
					for(x = 0; x < 2; x++)
					{
						ofmy = fmy*2 + y;
						ofmx = fmx*2 + x;
						outNeuIdx = sli*fmArea + ofmy*FMSIZE + ofmx;
						tmpVal = outNeu[outNeuIdx];
						if(tmpVal > max)
							max = tmpVal;
					}
				}
				outIdx = sli*outArea + fmy*FMSIZE/2 + fmx;
				outCPU[outIdx] = max;
			}
		}
	}
}


void initGPU()
{
	int outNeuVol = FILTNUM * FMSIZE * FMSIZE;  //512x32x32
	int outPolVol = FILTNUM * FMSIZE/2 * FMSIZE/2;  //512x16x16
	//int filtTensorVol = sizeof(short)*FILTNUM*FMDEPTH*FILTSIZE*FILTSIZE; //512x512x3x3
	int inNeuVol = sizeof(short)*FMDEPTH*FMSIZE*FMSIZE;	//512x32x32 
	int filtCOOVol = sizeof(short)*FILTNUM*FMDEPTH; //512x512x1

	//output from kernel 
	hipMalloc(&devoutNeu, sizeof(int)*outNeuVol);
	hipMalloc(&devPooling, sizeof(int)*outPolVol);
	
	//input to kernel
	hipMalloc(&devinNeu, inNeuVol);
	hipMemcpy(devinNeu, inNeu, inNeuVol, hipMemcpyHostToDevice);


	//input COO to kernel
	//hipMalloc(&devfiltCooNNZ, filtCOOVol);
	hipMalloc(&devfiltCooData, filtCOOVol);
	hipMalloc(&devfiltCooRow, filtCOOVol);
	hipMalloc(&devfiltCooCol, filtCOOVol);

	//hipMemcpy(devfiltCooNNZ, filtCooNNZ, filtCOOVol, hipMemcpyHostToDevice );
	hipMemcpy(devfiltCooData, filtCooData, filtCOOVol, hipMemcpyHostToDevice );
	hipMemcpy(devfiltCooRow, filtCooRow, filtCOOVol, hipMemcpyHostToDevice );
	hipMemcpy(devfiltCooCol, filtCooCol, filtCOOVol, hipMemcpyHostToDevice );

	//hipMemcpy(devoutNeu, outNeu,sizeof(int)*outNeuVol, hipMemcpyHostToDevice ); // debug for race outNeu
}


/***	Implement your CUDA Kernel here	***/
__global__
void convLayerGPU(short *InNeu, short *FiltCooData, short *FiltCooRow, short *FiltCooCol, int *outNeural, int *outPooling)
{
	int threadX = threadIdx.x + blockIdx.x * blockDim.x;
	int threadY = threadIdx.y + blockIdx.y * blockDim.y;
	int threadZ = threadIdx.z + blockIdx.z * blockDim.z;
	int ifmy, ifmx;
	int inNeuIdx, outNeuIdx, CooIdx;
	int fmArea = 1024;	//32x32
	int sum = 0;

	for (int i = 0; i < 512; ++i)
	{
		CooIdx = threadX*512 + i;

		ifmy = threadY - 3 / 2 + FiltCooRow[CooIdx];		
		ifmx = threadZ - 3 / 2 + FiltCooCol[CooIdx];		
		inNeuIdx = i * fmArea + ifmy * 32 + ifmx;	
		if(ifmy >= 0 && ifmy < 32 && ifmx >= 0 && ifmx < 32)	
			sum += FiltCooData[CooIdx] * InNeu[inNeuIdx];
	}


	outNeuIdx = threadX * fmArea + threadY*32 + threadZ;
	if(sum <= 0)
		outNeural[outNeuIdx] = 0;
	else
		outNeural[outNeuIdx] = sum;
}





__global__
void MaxPoolingGPU(int *outNeural, int *outPooling)  // Max Pooling with Window Size 2x2
{
	int threadX = threadIdx.x + blockIdx.x * blockDim.x;
	int threadY = threadIdx.y + blockIdx.y * blockDim.y;
	int threadZ = threadIdx.z + blockIdx.z * blockDim.z;

	int max, tmpVal, outNeuIdx, x, y;
	int fmArea = 1024;
	int outArea = 256;
	int  ofmy, ofmx, outIdx; // pooling varable

	outNeuIdx = threadX*fmArea + threadY*2*32 + threadZ*2;
	max = outNeural[outNeuIdx];
	for(y = 0; y < 2; y++)
	{
		for(x = 0; x < 2; x++)
		{
			ofmy = threadY*2 + y;
			ofmx = threadZ*2 + x;
			outNeuIdx = threadX*fmArea + ofmy*32 + ofmx;
			tmpVal = outNeural[outNeuIdx];
			if(tmpVal > max)
				max = tmpVal;
		}
	}
	outIdx = threadX*outArea + threadY*32/2 + threadZ;
	outPooling[outIdx] = max;
}


int main()
{
	float convLayerCPUExecTime, convLayerGPUExecTime;
	init();
	initCoo();
	int outSize = sizeof(int)*outputsize;

	timespec time_begin, time_end;
  	clock_gettime(CLOCK_REALTIME, &time_begin);
	convLayerCPU();
  	clock_gettime(CLOCK_REALTIME, &time_end);
	convLayerCPUExecTime = timespec_diff_us(time_begin, time_end);
	cout << " ================ Result ===================" << endl;
	cout << "CPU time for executing a typical convolutional layer = " <<  convLayerCPUExecTime / 1000 << "ms" << endl;



 	dim3 threadPerBlock(xThreadDim, yThreadDim, zThreadDim);
 	dim3 numBlocks(xDim/xThreadDim, yDim/yThreadDim, zDim/zThreadDim);
 	dim3 Pool_threadPerBlock(xThreadDim, yThreadDim, zThreadDim);
 	dim3 Pool_numBlocks(Pool_xDim/xThreadDim, Pool_yDim/yThreadDim, Pool_zDim/zThreadDim);

 	clock_gettime(CLOCK_REALTIME, &time_begin);
 	initGPU();


	convLayerGPU<<<numBlocks,threadPerBlock>>>(devinNeu , devfiltCooData, devfiltCooRow, devfiltCooCol, devoutNeu, devPooling);
	MaxPoolingGPU<<<Pool_numBlocks , Pool_threadPerBlock>>>(devoutNeu, devPooling);
	hipDeviceSynchronize();

	hipMemcpy(outGPU, devPooling, outSize, hipMemcpyDeviceToHost);


	clock_gettime(CLOCK_REALTIME, &time_end);
	convLayerGPUExecTime = timespec_diff_us(time_begin, time_end);
	cout << "GPU time for executing a typical convolutional layer = " << convLayerGPUExecTime / 1000 << "ms" << endl;


	//int OutSize = sizeof(int)*Outputsize;
	//hipMemcpy(outResult_neu, devoutNeu, OutSize, hipMemcpyDeviceToHost);

	//printf("BarrierSum = %d\n",outGlobalBarrier[0] );
	//printf("count = %d\n",outGlobalBarrier[1] );



	// check the Output of Neu
	/*for (int i = 0; i < 512*32*32; ++i)
		if (outNeu[i] == outResult_neu[i])
		{
			printf("wrong at =  %d \n", i);
			break;
		}
	printf("PASS!!!\n");*/
	// check the Output of GPU
	/*for (int i = 0; i < 512*16*16; ++i)
		if (outCPU[i] != outGPU[i])
		{
			printf("wrong at =  %d \n", i);
			break;
		}
	printf("PASS!!!\n");*/



	if(checker())
	{
		cout << "Congratulations! You pass the check." << endl;
		cout << "Speedup: " << (float)convLayerCPUExecTime / convLayerGPUExecTime << endl;
	}
	else
		cout << "Sorry! Your result is wrong." << endl;

	cout << "=====================================================" << endl;

	hipFree(&devoutNeu);
	hipFree(&devPooling);
	hipFree(&devinNeu);

	hipFree(&devfiltCooNNZ);
	hipFree(&devfiltCooData);
	hipFree(&devfiltCooRow);
	hipFree(&devfiltCooCol);

	hipFree(&devinNeuCooNNZ);
	hipFree(&devinNeuCooData);
	hipFree(&devinNeuCooRow);
	hipFree(&devinNeuCooCol);


	delete [] outResult;
	delete [] outResult_neu;
	ending();

	return 0;
}
